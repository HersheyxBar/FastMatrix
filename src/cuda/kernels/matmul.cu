extern "C" {


#include <hip/hip_runtime.h>
#include <stdio.h>

// Stub implementation that will be replaced with actual CUDA kernel
__attribute__((visibility("default")))
int matmul_kernel(
    const void* A,
    const void* B,
    void* C,
    unsigned int rowsA,
    unsigned int colsA,
    unsigned int colsB)
{
    return 0; // success
}

} // extern "C"